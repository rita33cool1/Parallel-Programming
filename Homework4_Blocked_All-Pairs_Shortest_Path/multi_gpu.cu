#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

const int INF = 1000000000;
const int BLOCK_SIZE = 32;
int* input(char *inFileName);

__global__ void ApspPhase1 (const int k_block, size_t pitch, int* const matrix) {
    __shared__ int temp_matrix[BLOCK_SIZE][BLOCK_SIZE];

    // initialize parameters
    const int x_id = threadIdx.x;
    const int y_id = threadIdx.y;
    int base_1D_addr = BLOCK_SIZE * k_block;
    const int k_row = base_1D_addr + y_id;
    const int k_column = base_1D_addr + x_id;

    // Load Data
    const int ori_id = k_row * pitch + k_column;
		temp_matrix[x_id][y_id] = matrix[ori_id];

    // Synchronize
    __syncthreads();

    // Relax Edge
    int new_dist;
    for (int u = 0; u < BLOCK_SIZE; ++u) {
        new_dist = temp_matrix[x_id][u] + temp_matrix[u][y_id];
        if (new_dist < temp_matrix[x_id][y_id]) {
            temp_matrix[x_id][y_id] = new_dist;
        }
        __syncthreads();
    }
		matrix[ori_id] = temp_matrix[x_id][y_id];
}

__global__ void ApspPhase2(const int k_block, size_t pitch, int* const matrix) {
	  if (blockIdx.x == k_block) return;

    // initialize parameters
    const int x_id = threadIdx.x;
    const int y_id = threadIdx.y;
    int base_1D_addr = BLOCK_SIZE * k_block;
    int k_row = base_1D_addr + y_id;
    int k_column = base_1D_addr + x_id;
    __shared__ int temp_ori_matrix[BLOCK_SIZE][BLOCK_SIZE];

    // Load Data
    int ori_id = k_row * pitch + k_column;
    temp_ori_matrix[y_id][x_id] = matrix[ori_id];

    if (blockIdx.y == 0) {
        k_column = BLOCK_SIZE * blockIdx.x + x_id;
    } else {
        k_row = BLOCK_SIZE * blockIdx.x + y_id;
    }
    __shared__ int temp_matrix[BLOCK_SIZE][BLOCK_SIZE];
    int ori_dist;
    ori_id = k_row * pitch + k_column;
    temp_matrix[y_id][x_id] = matrix[ori_id];
		ori_dist = matrix[ori_id];

    // Synchronize Data
    __syncthreads();

    // Relax Data
    int new_dist;
    if (blockIdx.y == 0) {
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            new_dist = temp_ori_matrix[y_id][u] + temp_matrix[u][x_id];
            if (new_dist < ori_dist) {
                ori_dist = new_dist;
            }
            temp_matrix[y_id][x_id] = ori_dist;
            __syncthreads();
        }
    } else {
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            new_dist = temp_matrix[y_id][u] + temp_ori_matrix[u][x_id];

            if (new_dist < ori_dist) {
                ori_dist = new_dist;
            }
            __syncthreads();
            temp_matrix[y_id][x_id] = ori_dist;
            __syncthreads();
        }
    }
    matrix[ori_id] = ori_dist;
}

__global__ void ApspPhase3(const int k_block, size_t pitch, int* const matrix) {
	  if (blockIdx.x == k_block || blockIdx.y == k_block) return;

    // Calculate addresses
    const int x_id = threadIdx.x;
    const int y_id = threadIdx.y;
    // blockDim.y is the number of elements in a row
    const int row_in_matrix = blockDim.y * blockIdx.y + y_id;
    // blockDim.x is the number of elements in a column
    const int col_in_matrix = blockDim.x * blockIdx.x + x_id;
    __shared__ int row_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int column_temp[BLOCK_SIZE][BLOCK_SIZE];
    int base_1D_addr = BLOCK_SIZE * k_block;
    int k_row = base_1D_addr + y_id;
    int k_column = base_1D_addr + x_id;

    // Load Data
    int ori_id;
    ori_id = k_row * pitch + col_in_matrix;
    row_temp[y_id][x_id] = matrix[ori_id];
    ori_id = row_in_matrix * pitch + k_column;
    column_temp[y_id][x_id] = matrix[ori_id];

    // Synchronize
    __syncthreads();

    // Edge Relax
    int ori_dist;
    int new_dist;
    ori_id = row_in_matrix * pitch + col_in_matrix;
    ori_dist = matrix[ori_id];
    for (int u = 0; u < BLOCK_SIZE; ++u) {
        new_dist = column_temp[y_id][u] + row_temp[u][x_id];
        if (ori_dist > new_dist) {
            ori_dist = new_dist;
        }
    }
   matrix[ori_id] = ori_dist;

}

int main(int argc, char* argv[]) {
  // Initial parameters
	int n, m;
	char* inFile;
	char* outFile;
	inFile = argv[1];
	outFile = argv[2];
	FILE* file = fopen(inFile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);

  // Build expanded adjacent matrix
	int* Dist;
	int block_size = BLOCK_SIZE;
  // Move the last and unused space in the final block to the last columan and the last row
	int expand = block_size - (n % block_size);
  int expand_dim = n + expand;
	Dist = (int * )malloc(expand_dim * expand_dim * sizeof(int));
	// printf("expand = %d\n", expand);
	for (int i = 0; i < expand_dim; i++){
			for (int j = 0; j < expand_dim; j++){
					if(i < n && j < n){
							if (i == j) {
									Dist[i*expand_dim+j] = 0;
							} else {
									Dist[i*expand_dim+j] = INF;
							}
					}
					else{
							Dist[i*expand_dim+j] = INF;
					}
			}
	}
	int pair[3];
	for (int i = 0; i < m; i++) {
			fread(pair, sizeof(int), 3, file);
			Dist[pair[0]*expand_dim + pair[1]] = pair[2];
	}
	fclose(file);
	 printf("%d\n", n);
	 for( int i = 0; i < expand_dim; i++){
	 		for( int j = 0; j < expand_dim; j++){
	 		}printf("\n");
	 }
  fflush(stdout);

  // Move data from host to device
	int* device_s;
	int* device_t;
	size_t height = expand_dim;
	size_t width = height * sizeof(int);
	size_t pitch;
  // Transfer 1D matrix into 2D matrix
	hipMallocPitch(&device_s, &pitch, width, height);
	hipMallocPitch(&device_t, &pitch, width, height);
	hipMemcpy2D(device_s, pitch, Dist, width, width, height, hipMemcpyHostToDevice);
	hipMemcpy2D(device_t, pitch, Dist, width, width, height, hipMemcpyHostToDevice);

  // Floyd Warshall
  int round = expand_dim / block_size;
	dim3 dimGrid1( 1, 1, 1);
	dim3 dimGrid2( round , 2 , 1);
	dim3 dimGrid3( round , round, 1);
  // threads in a block
	dim3 dimBlock(block_size, block_size, 1);
        //#pragma omp parallel num_threads(2)
	for( int i = 0; i < round; i++) {
      ApspPhase1<<<dimGrid1, dimBlock>>>(i, pitch/sizeof(int), device_s);
			ApspPhase2<<<dimGrid2, dimBlock>>>(i, pitch/sizeof(int), device_s);
			ApspPhase3<<<dimGrid3, dimBlock>>>(i, pitch/sizeof(int), device_s);
	}

  // Move data from device to host
	int* result = (int *) malloc(expand_dim*expand_dim*sizeof(int));
	hipMemcpy2D(result , width, device_s, pitch, width, height, hipMemcpyDeviceToHost);
	 for( int i = 0; i < expand_dim; i++){
	 		for( int j = 0; j < expand_dim; j++){
	 				//printf("%d ", result[i*expand_dim+j ]);
	 		}printf("\n");
	 }


  // Write file
	FILE *outfile = fopen(outFile, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
            if (result[i*expand_dim+j] >= INF)
                result[i*expand_dim+j] = INF;
		}
		fwrite(result+i*expand_dim, sizeof(int), n, outfile);
	}
  fclose(outfile);
	free(result);
	free(Dist);

	return 0;
}

